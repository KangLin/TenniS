#include "hip/hip_runtime.h"
#include <kernels/gpu/inner_prod.h>
#include <core/tensor_builder.h>
#include <kernels/cpu/math_cpu.h>
#include <global/operator_factory.h>
#include <backend/name.h>
#include <core/device.h>
#include <utils/assert.h>

#include ""
#include <hip/hip_runtime.h>
#include "kernels/gpu/cublas_device.h"
#include "core/device_context.h"
#include "utils/ctxmgr_lite.h"
#include "kernels/gpu/math_cublas.h"


namespace ts {
    namespace gpu {

        template<typename T>
        static __global__ void gpu_inner_prod_compute_run_kernel(int m, int n, int k, const T *A, const T *B, T *C) {
            __shared__ T ds_A[TRANS_BLOCK_DIM][TRANS_BLOCK_DIM];
            __shared__ T ds_B[TRANS_BLOCK_DIM][TRANS_BLOCK_DIM];

            int bx = blockIdx.x;
            int by = blockIdx.y;
            int tx = threadIdx.x;
            int ty = threadIdx.y;
            int Row = by * blockDim.y + ty;
            int Col = bx * blockDim.x + tx;

            T comp = 0;
            T Cvalue = 0;

            for (int t=0; t<(n - 1) / TRANS_BLOCK_DIM + 1; ++t) {
                if (Row < m && t * blockDim.x + tx < n)
                    ds_A[ty][tx] = A[Row*n+t*blockDim.x+tx];
                else
                    ds_A[ty][tx] = 0.0;

                if (t * blockDim.y + ty < n && Col < k)
                    ds_B[ty][tx] = B[(t*blockDim.y + ty)*k+Col];
                else
                    ds_B[ty][tx] = 0.0;

                __syncthreads();

                for (int i = 0; i < blockDim.x; ++i) {
                    //Cvalue += ds_A[ty][i] * ds_B[i][tx];
                    T t;
                    comp -= ds_A[ty][i] * ds_B[i][tx];
                    t = Cvalue - comp;
                    comp = (t - Cvalue) + comp;
                    Cvalue = t;
                }

                __syncthreads();

                if(Row < m && Col < k) {
                    C[Row*k+Col]=Cvalue;
                }
            }//end for
        
        
        }


        template<typename T>
        static void gpu_inner_prod_compute_run(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            const Shape &lhs_shape = lhs.sizes();
            const Shape &rhs_shape = rhs.sizes();

            const T *psrc = lhs.data<T>();
            const T *pdot = rhs.data<T>();
            T *pdst = out.data<T>();

#ifdef TS_USE_CUBLAS
            auto &context = ctx::ref<DeviceContext>();
            CublasDevice* handle = reinterpret_cast<CublasDevice*>(context.handle);
            auto cublas_handle = handle->get();

            cublas::math<T>::gemm(cublas_handle, cublas::NoTrans, cublas::NoTrans,
                lhs_shape[0], rhs_shape[1], lhs_shape[1], 1, psrc, pdot, 0, pdst);
            /*cublas::math<T>::gemm(cublas_handle,cublas::RowMajor,cublas::NoTrans, cublas::NoTrans, 
                lhs_shape[0], rhs_shape[1], lhs_shape[1], 1,psrc, lhs_shape[1], pdot, rhs_shape[1], 0,pdst, rhs_shape[1]);*/
            
#else
            dim3 blocksize(CUDA_BLOCK(rhs_shape[1], TRANS_BLOCK_DIM), CUDA_BLOCK(lhs_shape[0], TRANS_BLOCK_DIM),1);
            dim3 threadsize(TRANS_BLOCK_DIM, TRANS_BLOCK_DIM,1);
            gpu_inner_prod_compute_run_kernel<T> <<<blocksize, threadsize>>> (lhs_shape[0], lhs_shape[1], rhs_shape[1], psrc, pdot, pdst);

#endif
        }

        void InnerProd::inner_prod(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_inner_prod_compute_run<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
            default: {
                TS_LOG_ERROR << this->op() << " not support this data type: " << dtype << eject;
                break;
            }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(InnerProd, GPU, name::layer::inner_prod())

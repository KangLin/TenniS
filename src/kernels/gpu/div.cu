#include "hip/hip_runtime.h"
#include <kernels/gpu/div.h>
#include <core/tensor_builder.h>
#include <backend/name.h>
#include <utils/assert.h>
#include <global/operator_factory.h>
#include <core/device.h>

#include <numeric>
#include ""
#include <hip/hip_runtime.h>



namespace ts {
    namespace gpu {

        template<typename T>
        static __global__ void reduce_operator_scalar_kernel(T* data, int size, const T *scalar, T maxvalue, T minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                data[index] = (*scalar) == T(0)
                ? (data[index] > 0 ? maxvalue : minvalue)
                : data[index] / (*scalar);
            }
        }

        template<typename T>
        static __global__ void reduce_operator_same_shape_kernel(T* data, const T*bias, int size, T maxvalue, T minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                data[index] = (bias[index]) == T(0)
                ? (data[index] > 0 ? maxvalue : minvalue)
                : data[index] / (bias[index]);
            }
        }

        template<typename T>
        static __global__ void reduce_operator_bias_kernel(T* data, int size, int step, int slice,
                                        const T* bias, int biaslen, T maxvalue, T minvalue ) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                int dim = index % ( step * slice ) / (step);
                data[index] = (bias[dim]) == T(0)
                ? (data[index] > 0 ? maxvalue: minvalue)
                : data[index] / (bias[dim]);
            }
        }


        template<typename T>
        static __global__ void reduce_operator_kernel(T* out, int size, const T* lhs,  const T* rhs,
                                               int *lhsshape, int *lhsweight,
                                               int *rhsshape, int *rhsweight,
                                               int *outweight, int shapelen, T maxvalue, T minvalue) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= size)
                return;

            int *buffer1 = new int[shapelen];
            int *buffer2 = new int[shapelen];
            int i = 0;
            int k= 0;
            int *  ptmp;
            int *  ptr;

            ptr = buffer1;
            ptmp = outweight + 1;
            int ntmp = index;
            for(i= shapelen - 1; i; --i) {
                *ptr = ntmp / *ptmp;
                ntmp %= *ptmp;
                ++ptmp;
                ++ptr;
            }

            *ptr = ntmp;

            int lhsindex = 0;
            for(i=0; i<shapelen; ++i) {
                buffer2[i] = buffer1[i] % lhsshape[i];
            }

            for(k=0, i=1; i < shapelen; ++k,++i) {
                lhsindex += buffer2[k] * lhsweight[i];
            }
            lhsindex += buffer2[k];

            int rhsindex = 0;
            for(i=0; i<shapelen; ++i) {
                buffer2[i] = buffer1[i] % rhsshape[i];
            }

            for(k=0, i=1; i < shapelen; ++k,++i) {
                rhsindex += buffer2[k] * rhsweight[i];
            }
            rhsindex += buffer2[k];

            out[index] = (rhs[rhsindex]) == T(0)
                ? (lhs[lhsindex] > 0 ? maxvalue : minvalue)
                : lhs[lhsindex] / (rhs[rhsindex]);
            delete [] buffer1;
            delete [] buffer2;
        }


        template<typename T>
        static inline void div_gpu_compute_run(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            HypeShape lhs_hype(lhs.sizes());
            HypeShape rhs_hype(rhs.sizes());
            HypeShape out_hype(out.sizes());

            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            auto ncount = out.count();

            int *lhsshape = NULL;
            hipMalloc((void **)&lhsshape, lhs.sizes().size() * sizeof(int));

            int *rhsshape = NULL;
            hipMalloc((void **)&rhsshape, rhs.sizes().size() * sizeof(int));

            int *lhsweight = NULL;
            hipMalloc((void **)&lhsweight, lhs.sizes().size() * sizeof(int));

            int *rhsweight = NULL;
            hipMalloc((void **)&rhsweight, rhs.sizes().size() * sizeof(int));

            int *outweight = NULL;
            hipMalloc((void **)&outweight, out.sizes().size() * sizeof(int));


            hipMemcpy((void *)lhsshape, (void *)lhs.sizes().data(), lhs.sizes().size() * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy((void *)rhsshape, (void *)rhs.sizes().data(), rhs.sizes().size() * sizeof(int), hipMemcpyHostToDevice);

            hipMemcpy((void *)lhsweight, (void *)lhs_hype.weight().data(), lhs_hype.weight().size() * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy((void *)rhsweight, (void *)rhs_hype.weight().data(), rhs_hype.weight().size() * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy((void *)outweight, (void *)out_hype.weight().data(), out_hype.weight().size() * sizeof(int), hipMemcpyHostToDevice);

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();

            reduce_operator_kernel<T> <<< CUDA_BLOCK(ncount, CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, ncount,
                        plhs, prhs, lhsshape, lhsweight, rhsshape, rhsweight, outweight, out.sizes().size(),maxvalue, minvalue);

            hipFree(lhsshape);
            hipFree(rhsshape);

            hipFree(lhsweight);
            hipFree(rhsweight);
            hipFree(outweight);
        }


        template<typename T>
        static inline void div_gpu_compute_run_scalar(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();
            hipMemcpy((void *)pout, (void *)plhs, out.count() * sizeof(T), hipMemcpyDeviceToDevice);
            reduce_operator_scalar_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, out.count(), prhs,maxvalue, minvalue);

        }


        template<typename T>
        static inline void div_gpu_compute_run_same_shape(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();
            hipMemcpy((void *)pout, (void *)plhs, out.count() * sizeof(T), hipMemcpyDeviceToDevice);
            reduce_operator_same_shape_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, prhs, out.count(),maxvalue,minvalue);

        }


        template<typename T>
        static inline void div_gpu_compute_run_bias(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            auto &out_shape = out.sizes();
            auto number = std::accumulate(out_shape.begin(), out_shape.begin() + dim, 1, std::multiplies<int>());
            auto count = std::accumulate(out_shape.begin() + dim + 1, out_shape.end(), 1, std::multiplies<int>());

            auto channels = out_shape[dim];

            hipMemcpy((void *)pout, (void *)plhs, out.count() * sizeof(T), hipMemcpyDeviceToDevice);

            T maxvalue = std::numeric_limits<T>::max();
            T minvalue = std::numeric_limits<T>::lowest();
            reduce_operator_bias_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, out.count(),
                 count, channels, prhs, rhs.count(), maxvalue, minvalue);

        }


        void Div::reduce_with_broadcast(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "div not support this data type: " << dtype << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_scalar(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_scalar<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "div not support this data type: " << dtype << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_bias(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_bias<TYPE>(lhs, rhs, out, dim); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "div not support this data type: " << dtype << eject;
                    break;
                }
            }
        }

        void Div::reduce_with_same_shape(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { div_gpu_compute_run_same_shape<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "div not support this data type: " << dtype << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(Div, GPU, name::layer::div())


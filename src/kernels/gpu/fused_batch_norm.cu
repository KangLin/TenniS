#include "hip/hip_runtime.h"
#include <kernels/gpu/fused_batch_norm.h>
#include <core/tensor_builder.h>

#include <global/operator_factory.h>
#include <backend/name.h>
#include <utils/assert.h>
#include <core/device.h>
#include <vector>

#include ""
#include <hip/hip_runtime.h>



namespace ts {
    namespace gpu {

        template<typename T>
        static __global__ void gpu_fused_batch_norm_compute_kernel(T* data, int size, int step, int slice,
                                        const T* mean, const T* variance, const T* scale, const T* bias ) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                int dim = index % ( step * slice ) / (step);
                data[index] = (data[index] - mean[dim]) * variance[dim] * scale[dim] + bias[dim];
            }
        }



        template<typename T>
        static void gpu_fused_batch_norm_compute_run(const Tensor &x,
                                               const Tensor &mean, const Tensor &variance,
                                               const Tensor &scale, const Tensor &bias,
                                               int dim, float epsilon, Tensor &out) {
            const Shape &shape = x.sizes();
            //int predims = 1;
            int backdims = 1;
            //for (int i = 0; i < dim; i++) {
            //    predims *= shape[i];
            //}

            for (int i = dim + 1; i < shape.size(); i++) {
                backdims *= shape[i];
            }

            const T *psrc = x.data<T>();
            const T *pmean = mean.data<T>();
            const T *pvariance = variance.data<T>();
            const T *pscale = scale.data<T>();
            const T *pbias = bias.data<T>();
            T *pdst = out.data<T>();

            std::vector<T> vec(variance.count());
            hipMemcpy((void *)vec.data(), (void*)pvariance, vec.size() * sizeof(T), hipMemcpyDeviceToHost);

            for (int i = 0; i < vec.size(); i++) {
                vec[i] = T(1) / sqrt(vec[i] + T(epsilon));
            }

            T * pvar = NULL;
            hipMalloc((void **)&pvar, vec.size() * sizeof(T));
            hipMemcpy((void *)pvar, (void *)vec.data(),  vec.size() * sizeof(T), hipMemcpyHostToDevice);

            hipMemcpy((void *)pdst, (void *)psrc, out.count() * sizeof(T), hipMemcpyDeviceToDevice);

            gpu_fused_batch_norm_compute_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pdst, out.count(), backdims, shape[dim], pmean, pvar, pscale, pbias);

            hipFree(pvar);
  // only used in CPU
            //std::memcpy(pdst, psrc, out.count() * sizeof(T));

            //int stridedims = backdims * shape[dim];
            //int offset = 0;

            //std::vector<T> vec(variance.count());
            //for (int i = 0; i < vec.size(); i++) {
            //    vec[i] = T(1) / sqrt(pvariance[i] + T(epsilon));
            //}

            /*
            for (int i = 0; i < predims; i++) {
                for (int k = 0; k < shape[dim]; k++) {
                    offset = i * stridedims + k * backdims;
                    T mean_val = pmean[k];
                    T vec_val = vec[k];
                    T scale_val = pscale[k];
                    T bias_val = pbias[k];
                    T *pdst_temp = pdst + offset;
                    for (int m = backdims; m < backdims; m++) {
                        *pdst_temp = (*pdst_temp - mean_val) * vec_val * scale_val + bias_val;
                        pdst_temp++;
                    }
                }
            }
            */
        }

        void FusedBatchNorm::batch_norm(const Tensor &x, const Tensor &mean, const Tensor &variance,
                                   const Tensor &scale, const Tensor &bias,
                                   int dim, float epsilon, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_fused_batch_norm_compute_run<TYPE>(x, mean, variance, scale, bias, dim, epsilon, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support this data type: " << dtype << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(FusedBatchNorm, GPU, name::layer::fused_batch_norm())

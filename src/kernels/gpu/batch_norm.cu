#include "hip/hip_runtime.h"
#include <kernels/gpu/batch_norm.h>
#include <core/tensor_builder.h>

#include <global/operator_factory.h>
#include <backend/name.h>
#include <utils/assert.h>
#include <core/device.h>
#include <vector>

#include ""
#include <hip/hip_runtime.h>
#include <math_functions.hpp>


namespace ts {
    namespace gpu {
        template<typename T>
        static __global__ void gpu_batch_norm_compute_kernel(T* data, int size, int step, int slice,
                                        const T* mean, const T* variance ) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                int dim = index % ( step * slice ) / (step);
                data[index] = (data[index] - mean[dim]) / variance[dim];
            }
        }


        template<typename T>
        static void gpu_batch_norm_compute_run(const Tensor &x, const Tensor &mean,
                                               const Tensor &variance, int dim, float epsilon, Tensor &out) {
            const Shape &shape = x.sizes();
            //int predims = 1;
            int backdims = 1;
            //for (int i = 0; i < dim; i++) {
            //    predims *= shape[i];
            //}

            for (int i = dim + 1; i < shape.size(); i++) {
                backdims *= shape[i];
            }

            const T *psrc = x.data<T>();
            const T *pmean = mean.data<T>();
            const T *pvariance = variance.data<T>();
            T *pdst = out.data<T>();

            std::vector<T> vec(variance.count());

            memcpy((void*)vec.data(), MemoryDevice(CPU), vec.size() * sizeof(T),
                   (void*)pvariance, variance.device(), vec.size() * sizeof(T));


            for (int i = 0; i < vec.size(); i++) {
                vec[i] = T(1) / sqrt(vec[i] + T(epsilon));
            }

            T * pvar = nullptr;

            Shape tmpshape;
            tmpshape.resize(1);
            tmpshape[0] = vec.size();
            Tensor variance_tensor(variance.device(), variance.dtype(), tmpshape);
            pvar = variance_tensor.data<T>();

            memcpy((void*)pvar, variance.device(), vec.size() * sizeof(T),
                   (void*)vec.data(), MemoryDevice(CPU), vec.size() * sizeof(T));


            memcpy((void*)pdst, out.device(), out.count() * sizeof(T),
                   (void*)psrc, x.device(), x.count() * sizeof(T));

            gpu_batch_norm_compute_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pdst, out.count(), backdims, shape[dim], pmean, pvar);
           
        }


        void BatchNorm::batch_norm(const Tensor &x, const Tensor &mean, const Tensor &variance,
                                   int dim, float epsilon, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_batch_norm_compute_run<TYPE>(x, mean, variance, dim, epsilon, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << this->op() << " not support this data type: " << dtype << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(BatchNorm, GPU, name::layer::batch_norm())

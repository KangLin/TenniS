#include "hip/hip_runtime.h"
#include <kernels/gpu/add.h>
#include <core/tensor_builder.h>
#include <backend/name.h>
#include <utils/assert.h>
#include <global/operator_factory.h>
#include <core/device.h>

#include <numeric>

#include ""
#include <hip/hip_runtime.h>

//#ifdef TS_USE_OPENMP
//#include "kernels/common/openmp.h"
//#endif


namespace ts {
    namespace gpu {

        template<typename T>
        static __global__ void reduce_operator_scalar_kernel(T* data, int size, const T *scalar) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                data[index] += *scalar;
            }
        }

        template<typename T>
        static __global__ void reduce_operator_same_shape_kernel(T* data, const T*bias, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                //int dim = index % ( step * slice ) / (step);
                data[index] += bias[index];
            }
        }

        template<typename T>
        static __global__ void reduce_operator_bias_kernel(T* data, int size, int step, int slice,
                                        const T* bias, int biaslen ) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size) {
                int dim = index % ( step * slice ) / (step);
                data[index] += bias[dim];
            }
        }


        template<typename T>
        static __global__ void reduce_operator_kernel(T* out, int size, const T* lhs,  const T* rhs, 
                                               int *lhsshape, int *lhsweight,  
                                               int *rhsshape, int *rhsweight, 
                                               int *outweight, int shapelen) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= size) 
                return;

            int *buffer1 = new int[shapelen];
            int *buffer2 = new int[shapelen];
            int i = 0;
            int k= 0;
            int *  ptmp;
            int *  ptr;

            ptr = buffer1; 
            ptmp = outweight + 1;
            int ntmp = index;
            for(i= shapelen - 1; i; --i) {
                *ptr = ntmp / *ptmp;
                ntmp %= *ptmp;
                ++ptmp;
                ++ptr; 
            }

            *ptr = ntmp;

            int lhsindex = 0;
            for(i=0; i<shapelen; ++i) {
                buffer2[i] = buffer1[i] % lhsshape[i];    
            } 
                 
            for(k=0, i=1; i < shapelen; ++k,++i) {
                lhsindex += buffer2[k] * lhsweight[i]; 
            }
            lhsindex += buffer2[k];

            int rhsindex = 0;
            for(i=0; i<shapelen; ++i) {
                buffer2[i] = buffer1[i] % rhsshape[i];    
            } 
                 
            for(k=0, i=1; i < shapelen; ++k,++i) {
                rhsindex += buffer2[k] * rhsweight[i]; 
            }
            rhsindex += buffer2[k];
                
            out[index] = lhs[lhsindex] + rhs[rhsindex];

            delete [] buffer1;
            delete [] buffer2;
        }


        template<typename T>
        static inline void add_gpu_compute_run(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            HypeShape lhs_hype(lhs.sizes());
            HypeShape rhs_hype(rhs.sizes());
            HypeShape out_hype(out.sizes());

            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            auto ncount = out.count();

            int *lhsshape = NULL;
            hipMalloc((void **)&lhsshape, lhs.sizes().size() * sizeof(int));
            
            int *rhsshape = NULL;
            hipMalloc((void **)&rhsshape, rhs.sizes().size() * sizeof(int));

            int *lhsweight = NULL;
            hipMalloc((void **)&lhsweight, lhs.sizes().size() * sizeof(int));

            int *rhsweight = NULL;
            hipMalloc((void **)&rhsweight, rhs.sizes().size() * sizeof(int));

            int *outweight = NULL;
            hipMalloc((void **)&outweight, out.sizes().size() * sizeof(int));

            hipMemcpy((void *)lhsshape, (void *)lhs.sizes().data(), lhs.sizes().size() * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy((void *)rhsshape, (void *)rhs.sizes().data(), rhs.sizes().size() * sizeof(int), hipMemcpyHostToDevice);

            hipMemcpy((void *)lhsweight, (void *)lhs_hype.weight().data(), lhs_hype.weight().size() * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy((void *)rhsweight, (void *)rhs_hype.weight().data(), rhs_hype.weight().size() * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy((void *)outweight, (void *)out_hype.weight().data(), out_hype.weight().size() * sizeof(int), hipMemcpyHostToDevice);

            reduce_operator_kernel <<< CUDA_BLOCK(ncount, CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, ncount, 
                        plhs, prhs, lhsshape, lhsweight, rhsshape, rhsweight, outweight, out.sizes().size());

            hipFree(lhsshape);
            hipFree(rhsshape);

            hipFree(lhsweight);
            hipFree(rhsweight);
            hipFree(outweight);
        }


        template<typename T>
        static inline void add_gpu_compute_run_scalar(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();
            
            hipMemcpy((void *)pout, (void *)plhs, out.count() * sizeof(T), hipMemcpyDeviceToDevice);
            reduce_operator_scalar_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, out.count(), prhs);

        }


        template<typename T>
        static inline void add_gpu_compute_run_same_shape(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            hipMemcpy((void *)pout, (void *)plhs, out.count() * sizeof(T), hipMemcpyDeviceToDevice);
            reduce_operator_same_shape_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, prhs, out.count());

        }


        template<typename T>
        static inline void add_gpu_compute_run_bias(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            auto plhs = lhs.data<T>();
            auto prhs = rhs.data<T>();
            auto pout = out.data<T>();

            auto &out_shape = out.sizes();

            auto number = std::accumulate(out_shape.begin(), out_shape.begin() + dim, 1, std::multiplies<int>());
            auto count = std::accumulate(out_shape.begin() + dim + 1, out_shape.end(), 1, std::multiplies<int>());

            auto channels = out_shape[dim];

            hipMemcpy((void *)pout, (void *)plhs, out.count() * sizeof(T), hipMemcpyDeviceToDevice);

            reduce_operator_bias_kernel<T> <<< CUDA_BLOCK(out.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pout, out.count(), count, channels, prhs, rhs.count());

        }


        void Add::reduce_with_broadcast(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { add_gpu_compute_run<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "add not support this data type: " << dtype << eject;
                    break;
                }
            }
        }

        void Add::reduce_with_scalar(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { add_gpu_compute_run_scalar<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "add not support this data type: " << dtype << eject;
                    break;
                }
            }
        }

        void Add::reduce_with_bias(const Tensor &lhs, const Tensor &rhs, Tensor &out, int dim) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { add_gpu_compute_run_bias<TYPE>(lhs, rhs, out, dim); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "add not support this data type: " << dtype << eject;
                    break;
                }
            }
        }

        void Add::reduce_with_same_shape(const Tensor &lhs, const Tensor &rhs, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch(dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { add_gpu_compute_run_same_shape<TYPE>(lhs, rhs, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "add not support this data type: " << dtype << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(Add, GPU, name::layer::add())


#include "hip/hip_runtime.h"
#include <kernels/gpu/relu.h>
#include <algorithm>

#include "backend/name.h"
#include "global/operator_factory.h"
#include "kernels/gpu/memory_gpu.h"

#include <hip/hip_runtime.h>
#include <>

namespace ts {
    namespace gpu {

        template<typename T>
        __global__ static void relu_kernel(const T* input_data, T* output_data, int size) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index < size)
            {
                T val = input_data[index];
                output_data[index] = val > T(0.0) ? val : T(0.0);
            }
        }

        template<typename T>
        void cpu_relu_compute_run(const Tensor &x, Tensor &out) {
            const T *input_data = x.data<T>();
            T *output_data = out.data<T>();
            int count = out.count();
            // int bytes_num = count * sizeof(T);

            dim3 blockSize(CUDA_THREAD_NUM);
            dim3 gridSize(CUDA_BLOCK(count, blockSize.x));

            relu_kernel<T> << <gridSize, blockSize >> > (input_data, output_data, count);
        }

        void ReLU::active(const Tensor &x, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { cpu_relu_compute_run<TYPE>(x, out); break; }
                //DECLARE_COMPUTE_RUN(INT8, int8_t);
                //DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                //DECLARE_COMPUTE_RUN(INT16, int16_t);
                //DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                //DECLARE_COMPUTE_RUN(INT32, int32_t);
                //DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                //DECLARE_COMPUTE_RUN(INT64, int64_t);
                //DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
            default: {
                TS_LOG_ERROR << this->op() << " not support this data type: " << dtype << eject;
                break;
            }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(ReLU, ts::GPU, name::layer::relu())
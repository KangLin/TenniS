#include "hip/hip_runtime.h"
#include <kernels/gpu/add_bias.h>
#include <core/tensor_builder.h>
#include <global/operator_factory.h>
#include <backend/name.h>
#include <utils/assert.h>
#include <core/device.h>

#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "kernels/gpu/cuda_context.h"
#include "core/device_context.h"
#include "utils/ctxmgr_lite.h"

/////////////////////////////////////////////////
namespace ts {
    namespace gpu {
    template<typename T>
    static __global__ void add_bias_kernel(const T* base, T* data, int size, int step, int slice,
                                        const T* bias, int biaslen ) {
        int index = blockDim.x * blockIdx.x + threadIdx.x;
        if (index < size) {
            int dim = index % ( step * slice ) / (step);
            data[index] = base[index] + bias[dim];
        }
    }

    template<typename T>
    static void gpu_add_bias_compute_run(const Tensor &x, const Tensor &b, int dim, Tensor &out) {
        const Shape &shape = x.sizes();
        //int pre_dims = 1;
        int back_dims = 1;
        //for (int i = 0; i < dim; i++) {
        //    pre_dims *= shape[i];
        //}

        for (int i = dim + 1; i < shape.size(); i++) {
            back_dims *= shape[i];
        }

        const T *psrc = x.data<T>();
        const T *pbias = b.data<T>();
        T *pdst = out.data<T>();

        auto &context = ctx::ref<DeviceContext>();
        CUDAContextHandle* handle = reinterpret_cast<CUDAContextHandle*>(context.handle);
        auto hip_stream = handle->stream();

//        memcpy((void*)pdst, out.device(), x.count() * sizeof(T),
//               (void*)psrc, x.device(), x.count() * sizeof(T));


        add_bias_kernel<T> <<< CUDA_BLOCK(x.count(), CUDA_THREAD_NUM), CUDA_THREAD_NUM, 0, hip_stream >>> (psrc, pdst, x.count(), back_dims, shape[dim], pbias, b.count());

        //hipDeviceSynchronize();
    }

    void AddBias::add(const Tensor &x, const Tensor &b, int dim, Tensor &out) {
        // Notice: the all tensor' memory device are CPU, as given in running_memory_device
        DTYPE dtype = out.dtype();
        switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_add_bias_compute_run<TYPE>(x, b, dim, out); break; }
            DECLARE_COMPUTE_RUN(INT8, int8_t);
            DECLARE_COMPUTE_RUN(UINT8, uint8_t);
            DECLARE_COMPUTE_RUN(INT16, int16_t);
            DECLARE_COMPUTE_RUN(UINT16, uint16_t);
            DECLARE_COMPUTE_RUN(INT32, int32_t);
            DECLARE_COMPUTE_RUN(UINT32, uint32_t);
            DECLARE_COMPUTE_RUN(INT64, int64_t);
            DECLARE_COMPUTE_RUN(UINT64, uint64_t);
            DECLARE_COMPUTE_RUN(FLOAT32, float);
            DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
            default: {
                TS_LOG_ERROR << this->op() << " not support this data type: " << dtype << eject;
                break;
            }
        }
    }
  }
}
/////////////////////////////////////////////////

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(AddBias, GPU, name::layer::add_bias())


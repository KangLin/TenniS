#include "hip/hip_runtime.h"
#include <kernels/gpu/transpose.h>
#include <set>
#include <global/operator_factory.h>
#include <backend/name.h>
#include <core/device.h>
#include <utils/assert.h>
#include <core/tensor_builder.h>


#include ""
#include <hip/hip_runtime.h>



namespace ts {
    namespace gpu {

        template<typename T>
        static __global__ void Transpose_transpose_run_kernel(T* out, int size, const T* input,  
                                               int *inputshape, int *inputweight,
                                               int *outshape, int *outweight,
                                               int *permute, int shapelen) {
            int index = blockDim.x * blockIdx.x + threadIdx.x;
            if (index >= size)
                return;

            int *buffer1 = new int[shapelen];
            int *buffer2 = new int[shapelen];
            int i = 0;
            int k= 0;
            int *  ptmp;
            int *  ptr;

            ptr = buffer1;
            ptmp = inputweight + 1;
            int ntmp = index;
            for(i= shapelen - 1; i; --i) {
                *ptr = ntmp / *ptmp;
                ntmp %= *ptmp;
                ++ptmp;
                ++ptr;
            }

            *ptr = ntmp;

            for(i=0; i<shapelen; ++i) {
                buffer2[i] = buffer1[permute[i]];
            } 

            int outindex = 0;
            for(i=0; i<shapelen; ++i) {
                buffer1[i] = buffer2[i] % outshape[i];
            }

            for(k=0, i=1; i < shapelen; ++k,++i) {
                outindex += buffer1[k] * outweight[i];
            }
            outindex += buffer1[k];

            out[outindex] = input[index]; 

            delete [] buffer1;
            delete [] buffer2;
        }



        template<typename T>
        static void Transpose_transpose_run(
                const T *psrc, T *pdst, int len,
                const std::vector<int> &permute,
                const Shape &input_shape, const Shape &output_shape) {
            Shape tmpshape(input_shape.size());

            HypeShape hype_input_shape(input_shape);
            HypeShape hype_output_shape(output_shape);

            int *input_shape_dev = NULL;
            hipMalloc((void **)&input_shape_dev, input_shape.size() * sizeof(int));
            int *input_weight = NULL;
            hipMalloc((void **)&input_weight, input_shape.size() * sizeof(int));

            int *permute_shape_dev = NULL;
            hipMalloc((void **)&permute_shape_dev, permute.size() * sizeof(int));

            int *output_shape_dev = NULL;
            hipMalloc((void **)&output_shape_dev, output_shape.size() * sizeof(int));

            int *output_weight = NULL;
            hipMalloc((void **)&output_weight, output_shape.size() * sizeof(int));

            hipMemcpy((void *)input_shape_dev, (void *)input_shape.data(), input_shape.size() * sizeof(int), hipMemcpyHostToDevice);

            hipMemcpy((void *)input_weight, (void *)hype_input_shape.weight().data(), input_shape.size() * sizeof(int), hipMemcpyHostToDevice);

            hipMemcpy((void *)output_shape_dev, (void *)output_shape.data(), output_shape.size() * sizeof(int), hipMemcpyHostToDevice);

            hipMemcpy((void *)output_weight, (void *)hype_output_shape.weight().data(), output_shape.size() * sizeof(int), hipMemcpyHostToDevice);

            hipMemcpy((void *)permute_shape_dev, (void *)permute.data(), permute.size() * sizeof(int), hipMemcpyHostToDevice);

            Transpose_transpose_run_kernel<T> <<< CUDA_BLOCK(len, CUDA_THREAD_NUM), CUDA_THREAD_NUM >>> (pdst, len,
                        psrc, input_shape_dev, input_weight, output_shape_dev, output_weight, permute_shape_dev, input_shape.size());

            hipFree(input_shape_dev);
            hipFree(input_weight);
           
            hipFree(output_shape_dev);
            hipFree(output_weight);
            hipFree(permute_shape_dev);
        }

        template<typename T>
        static inline void gpu_transpose_compute_run(const Tensor &x, const std::vector<int> &permute, Tensor &out) {
            Transpose_transpose_run(x.data<T>(), out.data<T>(), x.count(), permute, x.sizes(), out.sizes());
        }

        void Transpose::transpose(const Tensor &x, const std::vector<int> &permute, Tensor &out) {
            // Notice: the all tensor' memory device are CPU, as given in running_memory_device
            DTYPE dtype = out.dtype();
            switch (dtype) {
#define DECLARE_COMPUTE_RUN(DTYPE, TYPE) \
        case DTYPE: { gpu_transpose_compute_run<TYPE>(x, permute, out); break; }
                DECLARE_COMPUTE_RUN(INT8, int8_t);
                DECLARE_COMPUTE_RUN(UINT8, uint8_t);
                DECLARE_COMPUTE_RUN(INT16, int16_t);
                DECLARE_COMPUTE_RUN(UINT16, uint16_t);
                DECLARE_COMPUTE_RUN(INT32, int32_t);
                DECLARE_COMPUTE_RUN(UINT32, uint32_t);
                DECLARE_COMPUTE_RUN(INT64, int64_t);
                DECLARE_COMPUTE_RUN(UINT64, uint64_t);
                DECLARE_COMPUTE_RUN(FLOAT32, float);
                DECLARE_COMPUTE_RUN(FLOAT64, double);
#undef DECLARE_COMPUTE_RUN
                default: {
                    TS_LOG_ERROR << "transpose not support this data type: " << dtype << eject;
                    break;
                }
            }
        }
    }
}

using namespace ts;
using namespace gpu;
TS_REGISTER_OPERATOR(Transpose, GPU, name::layer::transpose())
